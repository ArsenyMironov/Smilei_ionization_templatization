#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//
//! \file nvidiaParticles.cu
//
//! \brief contains the nvidiaParticles class methods
//! Extension of the Class Particles for GPU
//
// -----------------------------------------------------------------------------

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/sequence.h>


#include "Patch.h"
#include "gpu.h"
#include "nvidiaParticles.h"

// TODO(Etienne M): The makefile does not recognise this file and doesn't compute
// it's dependencies. If you make a modification in one of the header this file
// includes, you must `touch` this file. IF you dont do that you'll have ABI/ODR
// issues (!).

// Language: "in cell" means the number of cells for that, conversely, in cluster means 
// the number of clusters as a unit of length, etc.

////////////////////////////////////////////////////////////////////////////////
// Cell key manipulation functor definition
////////////////////////////////////////////////////////////////////////////////

//! Predicate for cell_keys
//! Return True if the entry is equal to `code`
template<int code>
struct cellKeyEquals
{
    constexpr __host__ __device__ bool
    operator()( const int& x ) const
    {
        return x == code;
    }
};

template<int key>
struct cellKeyBelow
{
    constexpr __host__ __device__ bool
    operator()( const int& x ) const
    {
        return x < key;
    }
};

namespace detail {

    ////////////////////////////////////////////////////////////////////////////////
    // Cluster manipulation functor definition
    ////////////////////////////////////////////////////////////////////////////////

    //! Cluster manipulation functionalities common to all dimension.
    //! NOTE: This only focus on GPU data manipulation. The host data shall
    //! not be handled here !
    //!
    struct Cluster
    {
    public:
        //! Same type as what is used in nvidia_cell_keys_
        //!
        using IDType         = int;
        using SizeType       = unsigned int;
        using DifferenceType = int;

    public:
        //! Compute the cell key for all the particles (not only a subset).
        //!
        static inline void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

        //! precondition:
        //!     - nvidia_cell_keys_ shall be sorted in non decreasing order
        //!     - last_index.data() is a pointer mapped to GPU via
        //!       HostDeviceMemoryManagement
        //!
        static inline void
        computeBinIndex( nvidiaParticles& particle_container );

        //! Sorting by cluster and binning
        //!
        //! precondition:
        //!     - particle_container is already sorted by cluster or
        //!       particle_container is not sorted anymore (after a push) but
        //!       still contains the old cluster key untouched.
        //!       PartBoundCond::apply will set the keys to zero !
        //!
        static inline void
        importAndSortParticles( nvidiaParticles& particle_container,
                                nvidiaParticles& particle_to_inject,
                                const Params&    parameters,
                                const Patch&     a_parent_patch );

    protected:
        template <typename InputIterator,
                  typename ClusterType>
        static void
        doComputeParticleClusterKey( InputIterator first,
                                     InputIterator last,
                                     ClusterType   cluster_type );

    };

    template <Cluster::DifferenceType kClusterWidth>
    struct Cluster1D : public Cluster
    {
    public:
        Cluster1D( double   inverse_x_cell_dimension,
                   SizeType local_x_dimension_in_cell,
        int CellStartingGlobalIndex_for_x);

        //! Compute the cell key of a_particle. a_particle shall be a tuple (from a
        //! zipiterator).
        //! The first value of a_particle is the cell key value, the other values are
        //! the positions x 
        template <typename Tuple>
        __host__ __device__ IDType
        Index( const Tuple& a_particle ) const;

        //! Compute the cell key of a particle range.
        //!
        static void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

        double   inverse_of_x_cell_dimension_;
        int CellStartingGlobalIndex_for_x_;
    };

    template <Cluster::DifferenceType kClusterWidth>
    struct Cluster2D : public Cluster
    {
    public:
        Cluster2D( double   inverse_x_cell_dimension,
                   double   inverse_y_cell_dimension,
                   SizeType local_x_dimension_in_cell,
                   SizeType local_y_dimension_in_cell,
                   int CellStartingGlobalIndex_for_x,
                   int CellStartingGlobalIndex_for_y);

        //! Compute the cell key of a_particle. a_particle shall be a tuple (from a
        //! zipiterator).
        //! The first value of a_particle is the cell key value, the other values are
        //! the positions x and y.
        //!
        template <typename Tuple>
        __host__ __device__ IDType
        Index( const Tuple& a_particle ) const;

        //! Compute the cell key of a particle range.
        //!
        static void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

    public:
        double   inverse_of_x_cell_dimension_;
        double   inverse_of_y_cell_dimension_;
        SizeType local_y_dimension_in_cluster_;
        int CellStartingGlobalIndex_for_x_;
        int CellStartingGlobalIndex_for_y_;
    };

    template <Cluster::DifferenceType kClusterWidth>
    struct Cluster3D : public Cluster
    {
    public:
        Cluster3D( double   inverse_x_cell_dimension,
                   double   inverse_y_cell_dimension,
                   double   inverse_z_cell_dimension,
                   SizeType local_x_dimension_in_cell,
                   SizeType local_y_dimension_in_cell,
                   SizeType local_z_dimension_in_cell,
                   int CellStartingGlobalIndex_for_x,
                   int CellStartingGlobalIndex_for_y,
                   int CellStartingGlobalIndex_for_z);

        //! Compute the cell key of a_particle. a_particle shall be a tuple (from a
        //! zipiterator).
        //! The first value of a_particle is the cell key value, the other values are
        //! the positions x and y.
        //!
        template <typename Tuple>
        __host__ __device__ IDType
        Index( const Tuple& a_particle ) const;

        //! Compute the cell key of a particle range.
        //!
        static void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

    public:
        double   inverse_of_x_cell_dimension_;
        double   inverse_of_y_cell_dimension_;
        double   inverse_of_z_cell_dimension_;
        SizeType local_y_dimension_in_cluster_;
        SizeType local_z_dimension_in_cluster_;
        int CellStartingGlobalIndex_for_x_;
        int CellStartingGlobalIndex_for_y_;
        int CellStartingGlobalIndex_for_z_;
    };


    //! This functor assign a cluster key to a_particle.
    //!
    template <typename ClusterType>
    class AssignClusterIndex
    {
    public:
        AssignClusterIndex( ClusterType cluster_type )
            : cluster_type_{ cluster_type }
        {
        }

        template <typename Tuple>
        __host__ __device__ void
        operator()( Tuple& a_particle ) const
        {
            thrust::get<0>( a_particle ) = cluster_type_.Index( a_particle ); //cluster key 
        }

    protected:
        ClusterType cluster_type_;
    };


    ////////////////////////////////////////////////////////////////////////////////
    // Cluster manipulation functor method definitions
    ////////////////////////////////////////////////////////////////////////////////

    inline void
    Cluster::computeParticleClusterKey( nvidiaParticles& particle_container,
                                        const Params&    parameters,
                                        const Patch&     a_parent_patch )
    {
        // This is where we do a runtime dispatch depending on the simulation's
        // dimensions.

        switch( particle_container.dimension() ) {
            case 1: {
                Cluster1D<Params::getGPUClusterWidth( 1 )>::computeParticleClusterKey( particle_container,
                                                                                                parameters,
                                                                                                a_parent_patch );
                break;
            }
            case 2: {
                Cluster2D<Params::getGPUClusterWidth( 2 )>::computeParticleClusterKey( particle_container,
                                                                                                parameters,
                                                                                                a_parent_patch );
                break;
            }
            case 3: {
                Cluster3D<Params::getGPUClusterWidth( 3 )>::computeParticleClusterKey( particle_container,
                                                                                                parameters,
                                                                                                a_parent_patch );
                break;
            }
            default:
                // Not implemented, only Cartesian 1D, 2D or 3D for the moment
                SMILEI_ASSERT( false );
                break;
        }
    }

    inline void
    Cluster::computeBinIndex( nvidiaParticles& particle_container )
    {
        SMILEI_GPU_ASSERT_MEMORY_IS_ON_DEVICE( particle_container.last_index.data() );

        Cluster::IDType* bin_upper_bound = smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( particle_container.last_index.data() );

        // SMILEI_ASSERT( thrust::is_sorted( thrust::device,
        //                                   static_cast<const IDType*>( particle_container.getPtrCellKeys() ),
        //                                   static_cast<const IDType*>( particle_container.getPtrCellKeys() ) + particle_container.deviceSize() ) );

        // NOTE: On some benchmark, I found this upper_bound usage faster than the counting_iterator (by a lot(!) ~x3, but
        // it's so fast anyway..)

        // thrust::upper_bound( thrust::device,
        //                      nvidia_cell_keys_.cbegin(), nvidia_cell_keys_.cend(),
        //                      key_bound_to_search.cbegin(), key_bound_to_search.cend(),
        //                      bin_upper_bound );

        // NOTE: A particle is in a bin if the index of the bin is the same integer value as the particle's cell key.
        // The particles are sorted by cell key. We can do a simple binary search to find the upper bound of a bin.
        //
        thrust::upper_bound( thrust::device,
                             static_cast<const IDType*>( particle_container.getPtrCellKeys() ),
                             static_cast<const IDType*>( particle_container.getPtrCellKeys() ) + particle_container.deviceSize(),
                             thrust::counting_iterator<Cluster::IDType>{ static_cast<Cluster::IDType>( 0 ) },
                             thrust::counting_iterator<Cluster::IDType>{ static_cast<Cluster::IDType>( particle_container.last_index.size() ) },
                             bin_upper_bound );

        // SMILEI_ASSERT( thrust::is_sorted( thrust::device,
        //                                   bin_upper_bound,
        //                                   bin_upper_bound + particle_container.last_index.size() ) );
    }

    inline void
    Cluster::importAndSortParticles( nvidiaParticles& particle_container,
                                     nvidiaParticles& particle_to_inject,
                                     const Params&    parameters,
                                     const Patch&     a_parent_patch )
    {
        const auto initial_count = particle_container.deviceSize();
        const auto inject_count  = particle_to_inject.deviceSize();

        // Locate out-of-bounds particles in array "available_places"
        const auto keys = particle_container.getPtrCellKeys();
        const auto erased_count = thrust::count_if( thrust::device, keys, keys + initial_count, cellKeyBelow<0>() );
        thrust::device_vector<int> available_places( erased_count );
        thrust::copy_if( thrust::device,
                         thrust::counting_iterator<int>{0},
                         thrust::counting_iterator<int>{ (int) initial_count },
                         keys,
                         available_places.begin(),
                         cellKeyBelow<0>() );
        
        const auto new_count = initial_count + inject_count - erased_count;
        
        // Copy the imported particles to available places
        particle_to_inject.scatterParticles( particle_container, available_places );
        // If there are more imported particles than places, copy the remaining imported particles at the end
        if( inject_count >= erased_count ) {
            particle_container.deviceResize( new_count );
            particle_container.pasteParticles( &particle_to_inject, initial_count, erased_count );
        // If there are more places than imported particles, the remaining places should be filled
        } else {
            const auto last_filled = available_places[inject_count];
            particle_container.eraseParticlesByPredicate( cellKeyBelow<0>(), last_filled );
            particle_container.deviceResize( new_count );
        }
        
        // Compute keys of particles
        computeParticleClusterKey( particle_container, parameters, a_parent_patch );
        
        // Sort particles by keys 
        // using particle_to_inject as a buffer (it is swapped with particle_container after sorting)
        particle_to_inject.deviceReserve( new_count ); // reserve a bit more memory for the final arrays
        particle_to_inject.deviceResize( new_count );
        particle_container.sortParticleByKey( particle_to_inject );
        
        // Recompute bin locations
        computeBinIndex( particle_container );
    }

    template <typename InputIterator,
              typename ClusterType>
    void
    Cluster::doComputeParticleClusterKey( InputIterator first,
                                          InputIterator last,
                                          ClusterType   cluster_type )
    {
        thrust::for_each( thrust::device,
                          first, last,
                          AssignClusterIndex<ClusterType>{ cluster_type } );
    }

    ////////////////////////////////////////////////////////////////////////////////
    // Cluster method definitions
    ////////////////////////////////////////////////////////////////////////////////

    template <Cluster::DifferenceType kClusterWidth>
    Cluster1D<kClusterWidth>::Cluster1D( double   inverse_x_cell_dimension,
                                         SizeType local_x_dimension_in_cell,
                                         int CellStartingGlobalIndex_for_x)
        : inverse_of_x_cell_dimension_{ inverse_x_cell_dimension }
        , CellStartingGlobalIndex_for_x_{CellStartingGlobalIndex_for_x}
    {
    }

    template <Cluster::DifferenceType kClusterWidth>
    Cluster2D<kClusterWidth>::Cluster2D( double   inverse_x_cell_dimension,
                                         double   inverse_y_cell_dimension,
                                         SizeType local_x_dimension_in_cell,
                                         SizeType local_y_dimension_in_cell,
                                         int CellStartingGlobalIndex_for_x, int CellStartingGlobalIndex_for_y )
        : inverse_of_x_cell_dimension_{ inverse_x_cell_dimension }
        , inverse_of_y_cell_dimension_{ inverse_y_cell_dimension }
        , local_y_dimension_in_cluster_{ local_y_dimension_in_cell / kClusterWidth }
        , CellStartingGlobalIndex_for_x_{CellStartingGlobalIndex_for_x}
        , CellStartingGlobalIndex_for_y_{CellStartingGlobalIndex_for_y}
    {
    }

    template <Cluster::DifferenceType kClusterWidth>
    Cluster3D<kClusterWidth>::Cluster3D( double   inverse_x_cell_dimension,
                                         double   inverse_y_cell_dimension,
                                         double   inverse_z_cell_dimension,
                                         SizeType local_x_dimension_in_cell,
                                         SizeType local_y_dimension_in_cell,
                                         SizeType local_z_dimension_in_cell,
                                         int CellStartingGlobalIndex_for_x,
                                         int CellStartingGlobalIndex_for_y, int CellStartingGlobalIndex_for_z )
        : inverse_of_x_cell_dimension_{ inverse_x_cell_dimension }
        , inverse_of_y_cell_dimension_{ inverse_y_cell_dimension }
        , inverse_of_z_cell_dimension_{ inverse_z_cell_dimension }
        , local_y_dimension_in_cluster_{ local_y_dimension_in_cell / kClusterWidth }
        , local_z_dimension_in_cluster_{ local_z_dimension_in_cell / kClusterWidth }
        , CellStartingGlobalIndex_for_x_{CellStartingGlobalIndex_for_x}
        , CellStartingGlobalIndex_for_y_{CellStartingGlobalIndex_for_y}
        , CellStartingGlobalIndex_for_z_{CellStartingGlobalIndex_for_z}
    {
    }

    template <Cluster::DifferenceType kClusterWidth>
    template <typename Tuple>
    __host__ __device__ typename Cluster1D<kClusterWidth>::IDType
    Cluster1D<kClusterWidth>::Index( const Tuple& a_particle ) const
    {
        const SizeType local_x_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<1>( a_particle ) *
                                                                                    inverse_of_x_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_x_;

        // These divisions will be optimized.
        // The integer division rounding behavior is expected.

        // NOTE: Flat tiles have been studied but were not as efficient for the
        // projection. The square provides the minimal perimeter (and thus ghost
        // cell amount) for a given area.
        static constexpr SizeType x_cluster_dimension_in_cell = kClusterWidth;

        const SizeType local_x_particle_cluster_coordinate_in_cluster = local_x_particle_coordinate_in_cell / x_cluster_dimension_in_cell;

        const SizeType cluster_index = local_x_particle_cluster_coordinate_in_cluster;

        return static_cast<IDType>( cluster_index );
    }

    template <Cluster::DifferenceType kClusterWidth>
    template <typename Tuple>
    __host__ __device__ typename Cluster2D<kClusterWidth>::IDType
    Cluster2D<kClusterWidth>::Index( const Tuple& a_particle ) const
    {
        const SizeType local_x_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<1>( a_particle ) *
                                                                                    inverse_of_x_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_x_;
        const SizeType local_y_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<2>( a_particle ) *
                                                                                    inverse_of_y_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_y_;

        // These divisions will be optimized.
        // The integer division rounding behavior is expected.

        // NOTE: Flat tiles have been studied but were not as efficient for the
        // projection. The square provides the minimal perimeter (and thus ghost
        // cell amount) for a given area.
        static constexpr SizeType x_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType y_cluster_dimension_in_cell = kClusterWidth;

        const SizeType local_x_particle_cluster_coordinate_in_cluster = local_x_particle_coordinate_in_cell / x_cluster_dimension_in_cell;
        const SizeType local_y_particle_cluster_coordinate_in_cluster = local_y_particle_coordinate_in_cell / y_cluster_dimension_in_cell;

        const SizeType y_stride = local_y_dimension_in_cluster_;

        // The indexing order is: x * ywidth * zwidth + y * zwidth + z
        const SizeType cluster_index = local_x_particle_cluster_coordinate_in_cluster * y_stride +
                                       local_y_particle_cluster_coordinate_in_cluster;

        return static_cast<IDType>( cluster_index );
    }
    
    template <Cluster::DifferenceType kClusterWidth>
    template <typename Tuple>
    __host__ __device__ typename Cluster3D<kClusterWidth>::IDType
    Cluster3D<kClusterWidth>::Index( const Tuple& a_particle ) const
    {
        const SizeType local_x_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<1>( a_particle ) *
                                                                                    inverse_of_x_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_x_;
        const SizeType local_y_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<2>( a_particle ) *
                                                                                    inverse_of_y_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_y_;
        const SizeType local_z_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<3>( a_particle ) *
                                                                                    inverse_of_z_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_z_;

        // These divisions will be optimized.
        // The integer division rounding behavior is expected.

        // NOTE: Flat tiles have been studied but were not as efficient for the
        // projection. The square provides the minimal perimeter (and thus ghost
        // cell amount) for a given area.
        static constexpr SizeType x_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType y_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType z_cluster_dimension_in_cell = kClusterWidth;

        const SizeType local_x_particle_cluster_coordinate_in_cluster = local_x_particle_coordinate_in_cell / x_cluster_dimension_in_cell;
        const SizeType local_y_particle_cluster_coordinate_in_cluster = local_y_particle_coordinate_in_cell / y_cluster_dimension_in_cell;
        const SizeType local_z_particle_cluster_coordinate_in_cluster = local_z_particle_coordinate_in_cell / z_cluster_dimension_in_cell;

        const SizeType y_stride = local_y_dimension_in_cluster_;
        const SizeType z_stride = local_z_dimension_in_cluster_;

        // The indexing order is: x * ywidth * zwidth + y * zwidth + z
        const SizeType cluster_index = local_x_particle_cluster_coordinate_in_cluster * z_stride * y_stride +
                                       local_y_particle_cluster_coordinate_in_cluster * z_stride +
                                       local_z_particle_cluster_coordinate_in_cluster;

        return static_cast<IDType>( cluster_index );
    }

    template <Cluster::DifferenceType kClusterWidth>
    void
    Cluster1D<kClusterWidth>::computeParticleClusterKey( nvidiaParticles& particle_container,
                                                         const Params&    parameters,
                                                         const Patch&     a_parent_patch )
    {
        const auto first = thrust::make_zip_iterator( thrust::make_tuple( particle_container.getPtrCellKeys(),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 0 ) ) ) );
        const auto last  = first + particle_container.deviceSize();
        int CellStartingGlobalIndex_for_x = a_parent_patch.getCellStartingGlobalIndex_noGC(0);
        doComputeParticleClusterKey( first, last,
                                     Cluster1D<Params::getGPUClusterWidth( 1 )>{ parameters.res_space[0],
                                                                                          parameters.patch_size_[0],
                                                                                          CellStartingGlobalIndex_for_x} );
    }

    template <Cluster::DifferenceType kClusterWidth>
    void
    Cluster2D<kClusterWidth>::computeParticleClusterKey( nvidiaParticles& particle_container,
                                                         const Params&    parameters,
                                                         const Patch&     a_parent_patch )
    {
        const auto first = thrust::make_zip_iterator( thrust::make_tuple( particle_container.getPtrCellKeys(),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 0 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 1 ) ) ) );
        const auto last  = first + particle_container.deviceSize();
        int CellStartingGlobalIndex_for_x = a_parent_patch.getCellStartingGlobalIndex_noGC(0);
        int CellStartingGlobalIndex_for_y = a_parent_patch.getCellStartingGlobalIndex_noGC(1);
        doComputeParticleClusterKey( first, last,
                                     Cluster2D<Params::getGPUClusterWidth( 2 )>{ parameters.res_space[0],
                                                                                          parameters.res_space[1],
                                                                                          parameters.patch_size_[0],
                                                                                          parameters.patch_size_[1],
                                                                                          CellStartingGlobalIndex_for_x,
                                                                                          CellStartingGlobalIndex_for_y } );
    }

    template <Cluster::DifferenceType kClusterWidth>
    void
    Cluster3D<kClusterWidth>::computeParticleClusterKey( nvidiaParticles& particle_container,
                                                         const Params&    parameters,
                                                         const Patch&     a_parent_patch )
    {
        const auto first = thrust::make_zip_iterator( thrust::make_tuple( particle_container.getPtrCellKeys(),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 0 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 1 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 2 ) ) ) );
        const auto last  = first + particle_container.deviceSize();
        int CellStartingGlobalIndex_for_x = a_parent_patch.getCellStartingGlobalIndex_noGC(0);
        int CellStartingGlobalIndex_for_y = a_parent_patch.getCellStartingGlobalIndex_noGC(1);
        int CellStartingGlobalIndex_for_z = a_parent_patch.getCellStartingGlobalIndex_noGC(2);
        doComputeParticleClusterKey( first, last,
                                     Cluster3D<Params::getGPUClusterWidth( 3 )>{ parameters.res_space[0],
                                                                                          parameters.res_space[1],
                                                                                          parameters.res_space[2],
                                                                                          parameters.patch_size_[0],
                                                                                          parameters.patch_size_[1],
                                                                                          parameters.patch_size_[2],
                                                                                          CellStartingGlobalIndex_for_x,
                                                                                          CellStartingGlobalIndex_for_y,
                                                                                          CellStartingGlobalIndex_for_z } );
    }

} // namespace detail


////////////////////////////////////////////////////////////////////////////////
// nvidiaParticles method definitions
////////////////////////////////////////////////////////////////////////////////

nvidiaParticles::nvidiaParticles( const Params& parameters,
                                  const Patch&  a_parent_patch )
    : Particles{}
    , parameters_{ &parameters }
    , parent_patch_{ &a_parent_patch }
    , gpu_nparts_{}
{
}

nvidiaParticles::~nvidiaParticles() {
    // Manage last_index if allocated on GPU
    if (smilei::tools::gpu::HostDeviceMemoryManagement::IsHostPointerMappedOnDevice( last_index.data() )) {
        smilei::tools::gpu::HostDeviceMemoryManagement::DeviceFree( last_index );
    }
}

void nvidiaParticles::deviceReserve( unsigned int particle_count, float growth_factor  )
{
    if( particle_count <= deviceCapacity() ) {
        // Dont reserve, for now we have enough capacity.
        return;
    }

    const unsigned int new_capacity = static_cast<unsigned int>( particle_count * growth_factor );

    for( auto prop: nvidia_double_prop_ ) {
        prop->reserve( new_capacity );
    }

    for( auto prop: nvidia_short_prop_ ) {
        prop->reserve( new_capacity );
    }

    if( tracked ) {
        nvidia_id_.reserve( new_capacity );
    }

    nvidia_cell_keys_.reserve( new_capacity );
}

void nvidiaParticles::deviceFree()
{
    for( auto prop: nvidia_double_prop_ ) {
        thrust::device_vector<double>().swap( *prop );
    }

    for( auto prop: nvidia_short_prop_ ) {
        thrust::device_vector<short>().swap( *prop );
    }

    if( tracked ) {
        thrust::device_vector<uint64_t>().swap( nvidia_id_ );
    }

    thrust::device_vector<int>().swap( nvidia_cell_keys_ );

    gpu_nparts_ = 0;
}

void nvidiaParticles::deviceResize( unsigned int new_size )
{
    for( auto prop: nvidia_double_prop_ ) {
        prop->resize( new_size );
    }

    for( auto prop: nvidia_short_prop_ ) {
        prop->resize( new_size );
    }

    if( tracked ) {
        nvidia_id_.resize( new_size );
    }
    
    nvidia_cell_keys_.resize( new_size );
    
    gpu_nparts_ = new_size;
}

// ---------------------------------------------------------------------------------------------------------------------
// Reset of Particles vectors
// Cell keys not affected
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::deviceClear()
{
    for( auto prop: nvidia_double_prop_ ) {
        prop->clear();
    }

    for( auto prop: nvidia_short_prop_ ) {
        prop->clear();
    }

    // TODO(Etienne M): Clear cell keys too ?

    if( tracked ) {
        nvidia_id_.clear();
    }
    
    gpu_nparts_ = 0;
}

// ---------------------------------------------------------------------------------------------------------------------
//! Reset cell_keys to default value
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::resetCellKeys(void)
{
    thrust::fill(nvidia_cell_keys_.begin(), nvidia_cell_keys_.begin() + gpu_nparts_, -1);
}

// -----------------------------------------------------------------------------
//! Initialize the particle properties on device as a mirror of the host definition
// -----------------------------------------------------------------------------
void nvidiaParticles::initializeDataOnDevice()
{
    SMILEI_ASSERT( Position.size() > 0 );
    // The world shall end if we call this function multiple times
    SMILEI_ASSERT( nvidia_double_prop_.empty() );

    // We sure that we have as many say, position dimension as the base class.
    nvidia_position_.resize( Position.size() );
    nvidia_momentum_.resize( 3 );

    // Initialize the list of pointers
    for( auto &pos: nvidia_position_ ) {
        nvidia_double_prop_.push_back( &pos );
    }
    for( auto &mom: nvidia_momentum_ ) {
        nvidia_double_prop_.push_back( &mom );
    }
    nvidia_double_prop_.push_back( &nvidia_weight_ );
    nvidia_short_prop_.push_back( &nvidia_charge_ );

    // Quantum parameter (for QED effects):
    // - if radiation reaction (continuous or discontinuous)
    // - if multiphoton-Breit-Wheeler if photons
    if( has_quantum_parameter ) {
        nvidia_double_prop_.push_back( &nvidia_chi_ );
    }

    // Optical Depth for Monte-Carlo processes:
    // - if the discontinuous (Monte-Carlo) radiation reaction
    // is activated, tau is the incremental optical depth to emission
    if( has_Monte_Carlo_process ) {
        nvidia_double_prop_.push_back( &nvidia_tau_ );
    }

    const auto hostParticleCount = Position[0].size();

    if( hostParticleCount == 0 ) {
        // Should we reserve some space ?
        // reserve( 100 );
    } else {
        copyFromHostToDevice();
    }

    if( prepareBinIndex() < 0 ) {
        // Either we deal with a simulation with unsupported space dimensions
        // (1D/AM) or we are not using OpenMP or we are dealing with particle
        // object without allocated bin (particle_to_move for instance).
        // We'll use the old, naive, unsorted particles injection
        // implementation.

        // Dont call setHostBinIndex. For particle that have binning this is a
        // redundant call. But for the particle that should not get binned
        // (ie: particle_to_move) , this is a bug (!) and will trigger an
        // assertion.

        // setHostBinIndex();
    } else {
        
        // At this point, a copy of the host particles and last_index is on the
        // device and we know we support the space dimension.
        detail::Cluster::computeParticleClusterKey( *this, *parameters_, *parent_patch_ );

        // The particles are not correctly sorted when created.
        sortParticleByKey();

        detail::Cluster::computeBinIndex( *this );
        setHostBinIndex();
    }
}

// -------------------------------------------------------------------------------------------------
//! Copy particle IDs from host to device
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::initializeIDsOnDevice()
{
    nvidia_id_.resize( Id.size() );
    thrust::copy((Id).begin(), (Id).end(), (nvidia_id_).begin());
}

// -------------------------------------------------------------------------------------------------
//! Copy the particles from host to device
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::copyFromHostToDevice()
{
    deviceResize( Position[0].size() );

    for( int idim = 0; idim < Position.size(); idim++ ) {
        thrust::copy( Position[idim].begin(), Position[idim].end(), nvidia_position_[idim].begin() );
    }

    for( int idim = 0; idim < Momentum.size(); idim++ ) {
        thrust::copy( Momentum[idim].begin(), Momentum[idim].end(), nvidia_momentum_[idim].begin() );
    }
    thrust::copy( Weight.begin(), Weight.end(), nvidia_weight_.begin() );

    thrust::copy( Charge.begin(), Charge.end(), nvidia_charge_.begin() );

    if( has_quantum_parameter ) {
        thrust::copy( Chi.begin(), Chi.end(), nvidia_chi_.begin() );
    }

    if( has_Monte_Carlo_process ) {
        thrust::copy( Tau.begin(), Tau.end(), nvidia_tau_.begin() );
    }

    if( tracked ) {
        thrust::copy( Id.begin(), Id.end(), nvidia_id_.begin() );
    }
}

// -------------------------------------------------------------------------------------------------
//! Copy device to host
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::copyFromDeviceToHost( bool copy_keys )
{
    for (int idim=0;idim<Position.size();idim++) {
        Position[idim].resize( gpu_nparts_ );
        thrust::copy((nvidia_position_[idim]).begin(), (nvidia_position_[idim]).begin()+gpu_nparts_, (Position[idim]).begin());
    }
    for (int idim=0;idim<Momentum.size();idim++) {
        Momentum[idim].resize( gpu_nparts_ );
        thrust::copy((nvidia_momentum_[idim]).begin(), (nvidia_momentum_[idim]).begin()+gpu_nparts_, (Momentum[idim]).begin());
    }
    Weight.resize( gpu_nparts_ );
    thrust::copy((nvidia_weight_).begin(), (nvidia_weight_).begin()+gpu_nparts_, (Weight).begin());
    Charge.resize( gpu_nparts_ );
    thrust::copy((nvidia_charge_).begin(), (nvidia_charge_).begin()+gpu_nparts_, (Charge).begin());
    if (has_quantum_parameter) {
        Chi.resize( gpu_nparts_ );
        thrust::copy((nvidia_chi_).begin(), (nvidia_chi_).begin()+gpu_nparts_, (Chi).begin());
    }
    if (has_Monte_Carlo_process) {
        Tau.resize( gpu_nparts_ );
        thrust::copy((nvidia_tau_).begin(), (nvidia_tau_).begin()+gpu_nparts_, (Tau).begin());
    }
    if (tracked) {
        Id.resize( gpu_nparts_ );
        thrust::copy((nvidia_id_).begin(), (nvidia_id_).begin()+gpu_nparts_, (Id).begin());
    }
    if (copy_keys) {
        cell_keys.resize( gpu_nparts_ );
        thrust::copy((nvidia_cell_keys_).begin(), (nvidia_cell_keys_).begin()+gpu_nparts_, (cell_keys).begin());
    }
}

unsigned int nvidiaParticles::deviceCapacity() const
{
    SMILEI_ASSERT( nvidia_momentum_.size() >= 1 );
    // Could be any particle component that we know will be used in any case.
    return nvidia_momentum_[0].capacity();
}

// -----------------------------------------------------------------------------
//! Move leaving particles to the buffer
// -----------------------------------------------------------------------------
void nvidiaParticles::copyLeavingParticlesToBuffer( Particles* buffer )
{
    copyParticlesByPredicate( buffer, cellKeyBelow<-1>() );
    buffer->copyFromDeviceToHost( true );
}


//! Copy particles which statisfy some predicate
template<typename Predicate>
void nvidiaParticles::copyParticlesByPredicate( Particles* buffer, Predicate pred )
{
    // Count particles satisfying the predicate
    const auto keys = getPtrCellKeys();
    const int nparts_to_copy = thrust::count_if( thrust::device, keys, keys + gpu_nparts_, pred );
    
    // Resize destination buffer (copy_if does not resize)
    nvidiaParticles* const dest = static_cast<nvidiaParticles*>( buffer );
    dest->deviceResize( nparts_to_copy );
    
    if( nparts_to_copy ) {
        // Copy the particles to the destination
        for( int ip = 0; ip < nvidia_double_prop_.size(); ip++ ) {
            const auto in = nvidia_double_prop_[ip]->begin();
            const auto out = dest->nvidia_double_prop_[ip]->begin();
            thrust::copy_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + gpu_nparts_, keys, out, pred );
        }
        for( int ip = 0; ip < nvidia_short_prop_.size(); ip++ ) {
            const auto in = nvidia_short_prop_[ip]->begin();
            const auto out = dest->nvidia_short_prop_[ip]->begin();
            thrust::copy_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + gpu_nparts_, keys, out, pred );
        }
        if( tracked ) {
            const auto in = nvidia_id_.begin();
            const auto out = dest->nvidia_id_.begin();
            thrust::copy_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + gpu_nparts_, keys, out, pred );
        }
        const auto in = nvidia_cell_keys_.begin();
        const auto out = dest->nvidia_cell_keys_.begin();
        thrust::copy_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + gpu_nparts_, keys, out, pred );
        SMILEI_ACCELERATOR_DEVICE_SYNC();
    }
}

int nvidiaParticles::addParticles( Particles* particles_to_inject )
{
    const auto nparts = gpu_nparts_;
    nvidiaParticles* to_inject = static_cast<nvidiaParticles*>( particles_to_inject );
    deviceResize( nparts + to_inject->gpu_nparts_ );
    pasteParticles( to_inject, nparts, 0 );
    return to_inject->gpu_nparts_;
}

void nvidiaParticles::pasteParticles( nvidiaParticles* particles_to_inject, size_t offset_in_output, size_t offset_in_input )
{
    const auto n = particles_to_inject->gpu_nparts_ - (int) offset_in_input;
    
    // Copy the particles to the destination
    for( int ip = 0; ip < nvidia_double_prop_.size(); ip++ ) {
        const auto in = particles_to_inject->nvidia_double_prop_[ip]->begin() + offset_in_input;
        const auto out = nvidia_double_prop_[ip]->begin() + offset_in_output;
        thrust::copy_n( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, n, out );
    }
    for( int ip = 0; ip < nvidia_short_prop_.size(); ip++ ) {
        const auto in = particles_to_inject->nvidia_short_prop_[ip]->begin() + offset_in_input;
        const auto out = nvidia_short_prop_[ip]->begin() + offset_in_output;
        thrust::copy_n( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, n, out );
    }
    if( tracked ) {
        const auto in = particles_to_inject->nvidia_id_.begin() + offset_in_input;
        const auto out = nvidia_id_.begin() + offset_in_output;
        thrust::copy_n( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, n, out );
    }
    SMILEI_ACCELERATOR_DEVICE_SYNC();
}

// -----------------------------------------------------------------------------
//! Erase `npart` particles from `ipart`
// -----------------------------------------------------------------------------
//void nvidiaParticles::eraseParticleOnDevice(int ipart, int npart) {
//
//    const auto first_particle = thrust::make_zip_iterator( thrust::make_tuple( std::begin( nvidia_position_[0] ),
//                                                                               std::begin( nvidia_momentum_[0] ),
//                                                                               std::begin( nvidia_momentum_[1] ),
//                                                                               std::begin( nvidia_momentum_[2] ),
//                                                                               std::begin( nvidia_weight_ ),
//                                                                               std::begin( nvidia_charge_ ) ) );
//
//    // Remove the other position values depending on the simulation's grid
//    // dimensions
//    for( int i = 1; i < position_dimension_count; ++i ) {
//        thrust::remove_if( thrust::device,
//                           std::begin( nvidia_position_[i] ),
//                           std::begin( nvidia_position_[i] ) + nparts,
//                           std::cbegin( nvidia_cell_keys_ ),
//                           cellKeyEquals<-1>() );
//    }
//
//}

// -----------------------------------------------------------------------------
//! Erase particles leaving the patch on device
// -----------------------------------------------------------------------------
int nvidiaParticles::eraseLeavingParticles()
{
    const auto nremoved = eraseParticlesByPredicate( cellKeyBelow<0>(), 0 );
    deviceResize( gpu_nparts_ - nremoved );
    return nremoved;
}

//! "Erase" particles but does not resize the arrays!
template<typename Predicate>
int nvidiaParticles::eraseParticlesByPredicate( Predicate pred, size_t offset )
{
    const auto keys = getPtrCellKeys();
    const int nparts_to_remove = thrust::count_if( thrust::device, keys + offset, keys + gpu_nparts_, pred );
    
    // Copy the particles to the destination
    // Using more memory, we could use the faster remove_copy_if
    // NOTE: remove_if is stable.
    for( auto prop: nvidia_double_prop_ ) {
        const auto in = prop->begin();
        thrust::remove_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in + offset, in + gpu_nparts_, keys + offset, pred );
    }
    for( auto prop: nvidia_short_prop_ ) {
        const auto in = prop->begin();
        thrust::remove_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in + offset, in + gpu_nparts_, keys + offset, pred );
    }
    if( tracked ) {
        const auto in = nvidia_id_.begin();
        thrust::remove_if( SMILEI_ACCELERATOR_ASYNC_POLYCY, in + offset, in + gpu_nparts_, keys + offset, pred );
    }
    SMILEI_ACCELERATOR_DEVICE_SYNC();

    return nparts_to_remove;
}


// ---------------------------------------------------------------------------------------------------------------------
//! Create n_additional_particles new particles at the end of vectors
//! Fill the new elements with 0
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::createParticles( int n_additional_particles )
{
    int n_particles = gpu_nparts_;
    int new_size = n_particles + n_additional_particles;
    
    deviceResize( new_size );
    
    for( auto prop: nvidia_double_prop_ ) {
         thrust::fill( prop->begin() + n_particles, prop->begin() + new_size, 0);
    }
    
    for( auto prop: nvidia_short_prop_ ) {
        thrust::fill( prop->begin() + n_particles, prop->begin() + new_size, 0);
    }
    
    if( tracked ) {
        thrust::fill( nvidia_id_.begin() + n_particles, nvidia_id_.begin() + new_size, 0 );
    }
    
    thrust::fill( nvidia_cell_keys_.begin() + n_particles, nvidia_cell_keys_.begin() + new_size, -1 );
}

//! Import Particles and sort depending if Binning is available or not
void nvidiaParticles::importAndSortParticles( Particles* particles_to_inject )
{
    if( parameters_->isGPUParticleBinningAvailable() ) {
        detail::Cluster::importAndSortParticles( *static_cast<nvidiaParticles*>( this ),
                                                 *static_cast<nvidiaParticles*>( particles_to_inject ),
                                                 *parameters_,
                                                 *parent_patch_ );
    } else {
        // When GPU particle binning is not supported, fallback to a naive implementation
        naiveImportAndSortParticles( static_cast<nvidiaParticles*>( particles_to_inject ) );
    }

    setHostBinIndex();
}

//! Sort by cell_keys_
//! This version synchronizes for every vector, but uses less buffers
void nvidiaParticles::sortParticleByKey()
{
    // Make a sorting map using the cell keys (like numpy.argsort)
    thrust::device_vector<int> index( gpu_nparts_ );
    thrust::sequence( thrust::device, index.begin(), index.end() );
    thrust::sort_by_key( thrust::device, nvidia_cell_keys_.begin(), nvidia_cell_keys_.end(), index.begin() );
    
    // Sort particles using thrust::gather, according to the sorting map
    thrust::device_vector<double> buffer( gpu_nparts_ );
    for( auto prop: nvidia_double_prop_ ) {
        thrust::gather( thrust::device, index.begin(), index.end(), prop->begin(), buffer.begin() );
        prop->swap( buffer );
    }
    buffer.clear();
    thrust::device_vector<short> buffer_short( gpu_nparts_ );
    for( auto prop: nvidia_short_prop_ ) {
        thrust::gather( thrust::device, index.begin(), index.end(), prop->begin(), buffer_short.begin() );
        prop->swap( buffer_short );
    }
    buffer_short.clear();
    if( tracked ) {
        thrust::device_vector<uint64_t> buffer_uint64( gpu_nparts_ );
        thrust::gather( thrust::device, index.begin(), index.end(), nvidia_id_.begin(), buffer_uint64.begin() );
        nvidia_id_.swap( buffer_uint64 );
        buffer_uint64.clear();
    }
}

//! Sort by cell_keys_
//! This version is asynchronous, but requires a buffer of equal size to be provided
void nvidiaParticles::sortParticleByKey( nvidiaParticles &buffer )
{
    // Make a sorting map using the cell keys (like numpy.argsort)
    thrust::device_vector<int> index( gpu_nparts_ );
    thrust::sequence( thrust::device, index.begin(), index.end() );
    thrust::sort_by_key( thrust::device, nvidia_cell_keys_.begin(), nvidia_cell_keys_.end(), index.begin() );
    
    // Sort particles using thrust::gather, according to the sorting map
    for( int ip = 0; ip < nvidia_double_prop_.size(); ip++ ) {
        thrust::gather( SMILEI_ACCELERATOR_ASYNC_POLYCY, index.begin(), index.end(), nvidia_double_prop_[ip]->begin(), buffer.nvidia_double_prop_[ip]->begin() );
    }
    for( int ip = 0; ip < nvidia_short_prop_.size(); ip++ ) {
        thrust::gather( SMILEI_ACCELERATOR_ASYNC_POLYCY, index.begin(), index.end(), nvidia_short_prop_[ip]->begin(), buffer.nvidia_short_prop_[ip]->begin() );
    }
    if( tracked ) {
        thrust::gather( SMILEI_ACCELERATOR_ASYNC_POLYCY, index.begin(), index.end(), nvidia_id_.begin(), buffer.nvidia_id_.begin() );
    }
    SMILEI_ACCELERATOR_DEVICE_SYNC();
    
    // Swap properties with their buffer
    for( int iprop = 0; iprop < nvidia_double_prop_.size(); iprop++ ) {
        nvidia_double_prop_[iprop]->swap( *buffer.nvidia_double_prop_[iprop] );
    }
    for( int iprop = 0; iprop < nvidia_short_prop_.size(); iprop++ ) {
        nvidia_short_prop_[iprop]->swap( *buffer.nvidia_short_prop_[iprop] );
    }
    if( tracked ) {
        nvidia_id_.swap( buffer.nvidia_id_ );
    }
}


void nvidiaParticles::scatterParticles( nvidiaParticles &dest, const thrust::device_vector<int> &index )
{
    const auto n = std::min( (int) index.size(), gpu_nparts_ );
    for( int ip = 0; ip < nvidia_double_prop_.size(); ip++ ) {
        const auto in = nvidia_double_prop_[ip]->begin();
        thrust::scatter( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + n, index.begin(), dest.nvidia_double_prop_[ip]->begin() );
    }
    for( int ip = 0; ip < nvidia_short_prop_.size(); ip++ ) {
        const auto in = nvidia_short_prop_[ip]->begin();
        thrust::scatter( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + n, index.begin(), dest.nvidia_short_prop_[ip]->begin() );
    }
    if( tracked ) {
        const auto in = nvidia_id_.begin();
        thrust::scatter( SMILEI_ACCELERATOR_ASYNC_POLYCY, in, in + n, index.begin(), dest.nvidia_id_.begin() );
    }
    SMILEI_ACCELERATOR_DEVICE_SYNC();
}

int nvidiaParticles::prepareBinIndex()
{
    if( first_index.size() == 0 ) {
        // Some Particles object do not have allocated bins, we skip theses.
        return -1;
    }

    const int kGPUBinCount = parameters_->getGPUBinCount();

    if( kGPUBinCount < 0 ) {
        // Unsupported space dimension or the offloading technology is not
        // supported, dont do GPU binning.
        return -1;
    }

    // We completely ignore/discard/overwrite what's done in
    // ParticleCreator::create regarding binning.
    // NOTE: maybe ParticleCreator::create should not be doing the particle
    // binning and should only be responsible for particle initialization (pos,
    // momentum etc.).
    // We are forced to deal with first_index even though its completely
    // redundant as long as the bins are dense (no holes).

    const auto particle_count = last_index.back();

    first_index.resize( 1 );
    last_index.resize( kGPUBinCount );

    // By definition it should be zero, so this is a redundant assignment
    first_index.back() = 0;

    // Dont try to allocate 2 times, even if it's harmless, that would be a bug!
    SMILEI_ASSERT( !smilei::tools::gpu::HostDeviceMemoryManagement::IsHostPointerMappedOnDevice( last_index.data() ) );

    // We'll need last_index to be on the GPU.

    // TODO(Etienne M): FREE. If we have load balancing or other patch
    // creation/destruction available (which is not the case on GPU ATM),
    // we should be taking care of freeing this GPU memory.
    smilei::tools::gpu::HostDeviceMemoryManagement::DeviceAllocate( last_index );

    return 0;
}

void nvidiaParticles::setHostBinIndex()
{
    // TODO(Etienne M): You may want to inject, create etc. into a non binned
    // nvidiaParticles object (without allocated first/last_index). For now, we
    // assert it does not happen. I think a fix only requires:
    //  if( last_index.empty() ) { return; }
    //
    SMILEI_ASSERT( !last_index.empty() );

    last_index.back() = deviceSize();
    last_index[0]     = last_index.back();
}

void nvidiaParticles::naiveImportAndSortParticles( nvidiaParticles* particles_to_inject )
{
    // Erase particles that leaves this patch
    eraseLeavingParticles();

    // Inject newly arrived particles in particles_to_inject
    const size_t current_size = gpu_nparts_;
    deviceResize( current_size + particles_to_inject->size() );
    pasteParticles( particles_to_inject, current_size, 0 );
    particles_to_inject->clear();
}

extern "C"
{
    void* CreateGPUParticles( const void* parameters, const void* a_parent_patch )
    {
        return new nvidiaParticles{ *static_cast<const Params*>( parameters ),
                                    *static_cast<const Patch*>( a_parent_patch ) };
    }
}
